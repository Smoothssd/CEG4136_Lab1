
#include <hip/hip_runtime.h>
//#include <GL/glut.h>
//#include <vector>
//#include <iostream>
//#include <random>    // Pour std::shuffle et std::mt19937 // For std::shuffle and std::mt19937
//#include <algorithm> // Pour std::shuffle // For std::shuffle
//#include <stdlib.h>
//#include <stdio.h>
//#include <time.h>
//#include <chrono>
//
//#include "cuda.h"
//#include "cuda_runtime.h"
//#include "curand_kernel.h"
//#include "device_launch_parameters.h"
//
//#define N 1000  // Taille de la grille // Grid size
//#define BURN_DURATION 5000  // Dur�e de combustion d'un arbre en millisecondes (5 secondes) // Tree burning duration in milliseconds (5 seconds)
//#define FIRE_START_COUNT 100  // Nombre initial d'incendies // Initial number of fire locations
//
//// Utilisation de vecteurs pour g�rer la m�moire // Using vectors to manage memory
//// use one-dimension vector
//bool useCUDA = false;
//
//std::vector<int> forest(N* N, 0);
//std::vector<int> burnTime(N* N, 0);
//
//int simulationDuration = 60000;  // Dur�e de la simulation (60 secondes) // Simulation duration (60 seconds)
//int startTime = 0;  // Temps de d�part en millisecondes // Start time in milliseconds
//int elapsedTime = 0;  // Temps �coul� // Elapsed time
//float spreadProbability = 0.3f;  // Probabilit� que le feu se propage � un arbre voisin // Probability that fire spreads to a neighboring tree
//
//bool isPaused = false;  // Indicateur de pause // Pause indicator
//int pauseStartTime = 0;  // Temps de d�but de la pause // Start time of pause
//
//float zoomLevel = 1.0f;  // Niveau de zoom // Zoom level
//float offsetX = 0.0f, offsetY = 0.0f;  // D�calage horizontal et vertical pour le d�placement // Horizontal and vertical offset for movement
//float moveSpeed = 0.05f;  // Vitesse de d�placement de la vue // View movement speed
//
//bool dragging = false;  // Indicateur de glisser-d�poser avec la souris // Mouse drag indicator
//int lastMouseX, lastMouseY;  // Derni�re position de la souris lors du clic // Last mouse position when clicked
//
//// Device pointers
//int* d_forest = nullptr;
//int* d_burnTime = nullptr;
//int* d_new_forest = nullptr;
//int* d_new_burnTime = nullptr;
//
////// Define global curandState array
////curandState* d_states = nullptr;
////
////// Kernel to initialize curandState array
////__global__ void initCurandStates(curandState* states, unsigned long seed, int d_N) {
////    int idx = blockIdx.x * blockDim.x + threadIdx.x;
////    int jdx = blockIdx.y * blockDim.y + threadIdx.y;
////    if (idx < d_N && jdx < d_N) {
////        int index = idx * d_N + jdx;
////        curand_init(seed, index, 0, &states[index]);
////    }
////}
//
//
//// Function to check CUDA errors
//#define cudaCheckError() {                                           \
//    cudaError_t e=cudaGetLastError();                                \
//    if(e!=cudaSuccess) {                                             \
//        printf("CUDA Error %s:%d: %s\n", __FILE__, __LINE__,         \
//                cudaGetErrorString(e));                              \
//        exit(EXIT_FAILURE);                                          \
//    }                                                                \
//}
//
//// CUDA Kernel to update the forest
//__global__ void updateForestKernel(int* d_forest, int* d_burnTime, int* d_new_forest, int* d_new_burnTime, int d_N, float spreadProbability, curandState* states) {
//    int idx = blockIdx.x * blockDim.x + threadIdx.x;
//    int jdx = blockIdx.y * blockDim.y + threadIdx.y;
//
//    if (idx < d_N && jdx < d_N) {
//        int index = idx * d_N + jdx;
//
//        // optimize random state
//        curandState localState = states[index];
//        // Initialize new forest and burnTime with current values
//        d_new_forest[index] = d_forest[index];
//        d_new_burnTime[index] = d_burnTime[index];
//
//        if (d_forest[index] == 2) {         // If the tree is on fire
//            d_new_burnTime[index] -= 200;   // Decrease burning time by 200 ms
//
//            if (d_new_burnTime[index] <= 0) {
//                d_new_forest[index] = 3;    // Mark as burned
//                d_new_burnTime[index] = 0;
//            }
//            else {
//
//                // Spread fire to neighbors with probability
//                // up
//                if (idx > 0) {
//                    int neighbor = (idx - 1) * d_N + jdx;
//                    if (d_forest[neighbor] == 1) {  // If neighbor has a tree
//                        float randVal = curand_uniform(&localState);
//                        if (randVal < spreadProbability) {
//                            d_new_forest[neighbor] = 2;  // Ignite the neighbor tree
//                            d_new_burnTime[neighbor] = BURN_DURATION;  // Set burn duration
//                        }
//                    }
//                }
//                // down
//                if (idx < d_N - 1) {
//                    int neighbor = (idx + 1) * d_N + jdx;
//                    if (d_forest[neighbor] == 1) {
//                        float randVal = curand_uniform(&localState);
//                        if (randVal < spreadProbability) {
//                            d_new_forest[neighbor] = 2;
//                            d_new_burnTime[neighbor] = BURN_DURATION;
//                        }
//                    }
//                }
//                // left
//                if (jdx > 0) {
//                    int neighbor = idx * d_N + (jdx - 1);
//                    if (d_forest[neighbor] == 1) {
//                        float randVal = curand_uniform(&localState);
//                        if (randVal < spreadProbability) {
//                            d_new_forest[neighbor] = 2;
//                            d_new_burnTime[neighbor] = BURN_DURATION;
//                        }
//                    }
//                }
//                // right
//                if (jdx < d_N - 1) {
//                    int neighbor = idx * d_N + (jdx + 1);
//                    if (d_forest[neighbor] == 1) {
//                        float randVal = curand_uniform(&localState);
//                        if (randVal < spreadProbability) {
//                            d_new_forest[neighbor] = 2;
//                            d_new_burnTime[neighbor] = BURN_DURATION;
//                        }
//                    }
//                }
//            }
//        }
//    }
//}
//
//// Fonction pour initialiser la for�t // Function to initialize the forest
//void initializeForest() {
//
//    // Initialisation de la for�t avec 50% d'arbres // Initializing the forest with 50% trees
//    for (int i = 0; i < N; i++) {
//        for (int j = 0; j < N; j++) {
//            forest[i * N + j] = rand() % 2;  // 50% d'arbres (1), 50% vide (0) // 50% trees (1), 50% empty space (0)
//            burnTime[i * N + j] = 0;  // Aucun arbre ne br�le au d�part // No tree is burning at the start
//        }
//    }
//
//    // Liste de positions disponibles pour allumer les feux // List of available positions to start fires
//    std::vector<std::pair<int, int>> availablePositions;
//    for (int i = 0; i < N; i++) {
//        for (int j = 0; j < N; j++) {
//            if (forest[i * N + j] == 1) {  // Ajouter les positions avec des arbres dans la liste // Add positions with trees to the list
//                availablePositions.push_back({ i, j });
//            }
//        }
//    }
//
//    // M�langer les positions disponibles pour une distribution plus uniforme // Shuffle the available positions for a more uniform distribution
//    std::random_device rd;  // G�n�rateur de nombres al�atoires bas� sur l'impl�mentation du syst�me // Random number generator based on system implementation
//    std::mt19937 g(rd());   // G�n�rateur de nombres pseudo-al�atoires bas� sur Mersenne Twister // Mersenne Twister-based pseudo-random number generator
//    std::shuffle(availablePositions.begin(), availablePositions.end(), g);
//
//    // Allumer des feux de mani�re uniforme sur la grille // Ignite fires uniformly across the grid
//    for (int fire = 0; fire < FIRE_START_COUNT && !availablePositions.empty(); fire++) {
//        int fireX = availablePositions[fire].first;
//        int fireY = availablePositions[fire].second;
//
//        forest[fireX * N + fireY] = 2;  // Allumer l'arbre en feu // Ignite the tree
//        burnTime[fireX * N + fireY] = BURN_DURATION;  // D�finir le temps de combustion // Set the burn duration
//    }
//
//    startTime = glutGet(GLUT_ELAPSED_TIME);  // R�initialiser le temps de d�part // Reset start time
//    elapsedTime = 0;  // R�initialiser le temps �coul� // Reset elapsed time
//    isPaused = false;  // Fin de la pause // End of pause
//
//
//    // Allocate device memory if not already allocated
//    if (d_forest == nullptr) {
//        cudaMalloc((void**)&d_forest, N * N * sizeof(int));
//        cudaCheckError();
//    }
//    if (d_burnTime == nullptr) {
//        cudaMalloc((void**)&d_burnTime, N * N * sizeof(int));
//        cudaCheckError();
//    }
//    if (d_new_forest == nullptr) {
//        cudaMalloc((void**)&d_new_forest, N * N * sizeof(int));
//        cudaCheckError();
//    }
//    if (d_new_burnTime == nullptr) {
//        cudaMalloc((void**)&d_new_burnTime, N * N * sizeof(int));
//        cudaCheckError();
//    }
//    if (d_states == nullptr) {
//        cudaMalloc((void**)&d_states, N * N * sizeof(curandState));
//    }
//    // Copy initial forest and burnTime to device
//    cudaMemcpy(d_forest, forest.data(), N * N * sizeof(int), cudaMemcpyHostToDevice);
//    cudaCheckError();
//    cudaMemcpy(d_burnTime, burnTime.data(), N * N * sizeof(int), cudaMemcpyHostToDevice);
//    cudaCheckError();
//
//}
//
//// Fonction d'initialisation OpenGL // OpenGL initialization function
//void initGL() {
//    glClearColor(1.0, 1.0, 1.0, 1.0);  // Couleur de fond blanche // White background color
//    glEnable(GL_DEPTH_TEST);  // Activer le test de profondeur // Enable depth test
//}
//
//// Fonction pour dessiner la grille // Function to draw the grid
//void drawForest() {
//    float cellSize = 2.0f / N;  // Taille de chaque cellule ajust�e par la taille N // Adjusted cell size based on grid size N
//
//    for (int i = 0; i < N; i++) {
//        for (int j = 0; j < N; j++) {
//            // Choisir la couleur en fonction de l'�tat de la cellule // Set color based on the state of the cell
//            if (forest[i * N + j] == 0 && burnTime[i * N + j] == 0) {
//                glColor3f(0.8f, 0.8f, 0.8f);  // Espace vide (gris) // Empty space (gray)
//            }
//            else if (forest[i * N + j] == 1) {
//                glColor3f(0.0f, 1.0f, 0.0f);  // Arbre (vert) // Tree (green)
//            }
//            else if (forest[i * N + j] == 2) {
//                glColor3f(1.0f, 0.0f, 0.0f);  // Arbre en feu (rouge) // Tree on fire (red)
//            }
//            else if (forest[i * N + j] == 3) {
//                glColor3f(0.0f, 0.0f, 0.0f);  // Arbre br�l� (noir) // Burned tree (black)
//            }
//
//            // Dessiner la cellule // Draw the cell
//            float x = -1.0f + j * cellSize;
//            float y = -1.0f + i * cellSize;
//            glBegin(GL_QUADS);
//            glVertex2f(x, y);
//            glVertex2f(x + cellSize, y);
//            glVertex2f(x + cellSize, y + cellSize);
//            glVertex2f(x, y + cellSize);
//            glEnd();
//        }
//    }
//}
//
//// use cuda to update the forest and fire propagation
//void updateForestCUDA() {
//
//    // create cuda event
//    cudaEvent_t start, middle, stop;
//    cudaEventCreate(&start);
//    cudaEventCreate(&middle);
//    cudaEventCreate(&stop);
//
//    // record cuda event
//    cudaEventRecord(start, 0);
//
//
//    // Define CUDA block and grid sizes
//    dim3 threadsPerBlock(16, 16);
//    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
//        (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
//
//    unsigned long seed = time(NULL);  // Use current time as seed
//
//    // init random
//    if (d_states == nullptr) {
//        cudaMalloc(&d_states, N * N * sizeof(curandState));
//        cudaCheckError();
//    }
//    initCurandStates << <numBlocks, threadsPerBlock >> > (d_states, seed, N);
//    cudaCheckError();
//
//    // Launch the CUDA kernel to update the forest
//    updateForestKernel << <numBlocks, threadsPerBlock >> > (d_forest, d_burnTime, d_new_forest, d_new_burnTime, N, spreadProbability, d_states);
//    cudaCheckError();
//    cudaDeviceSynchronize();
//    cudaCheckError();
//
//    cudaEventRecord(middle, 0);
//    cudaEventSynchronize(middle);
//    float middleTime;
//    cudaEventElapsedTime(&middleTime, start, middle);
//    printf("CUDA middleTime took %.3f ms\n", middleTime);
//
//    // Copy new_forest and new_burnTime back to d_forest and d_burnTime
//    cudaMemcpy(d_forest, d_new_forest, N * N * sizeof(int), cudaMemcpyDeviceToDevice);
//    cudaCheckError();
//    cudaMemcpy(d_burnTime, d_new_burnTime, N * N * sizeof(int), cudaMemcpyDeviceToDevice);
//    cudaCheckError();
//
//    // Copy the updated forest and burnTime back to host for rendering
//    cudaMemcpy(forest.data(), d_forest, N * N * sizeof(int), cudaMemcpyDeviceToHost);
//    cudaCheckError();
//    cudaMemcpy(burnTime.data(), d_burnTime, N * N * sizeof(int), cudaMemcpyDeviceToHost);
//    cudaCheckError();
//
//    // Check if all fires are out
//    bool allBurnedOut = true;
//    for (int i = 0; i < N; i++) {
//        for (int j = 0; j < N; j++)
//            if (forest[i * N + j] == 2) {
//                allBurnedOut = false;
//                break;
//            }
//    }
//
//    if (allBurnedOut) {  // Si tous les feux sont �teints, mettre la simulation en pause // If all fires are out, pause the simulation
//        isPaused = true;
//        pauseStartTime = glutGet(GLUT_ELAPSED_TIME);
//    }
//
//    // end cuda event
//    cudaEventRecord(stop, 0);
//    cudaEventSynchronize(stop);
//
//    // calculate time
//    float elapsedTime;
//    cudaEventElapsedTime(&elapsedTime, start, stop);
//    printf("CUDA updateForest took %.3f ms\n", elapsedTime);
//
//    cudaEventDestroy(start);
//    cudaEventDestroy(stop);
//}
//
//// Fonction pour mettre � jour la for�t et la propagation du feu // Function to update the forest and fire propagation
//void updateForestSequential() {
//    auto start = std::chrono::high_resolution_clock::now();
//
//    std::vector<int> newForest = forest;  // Copie la for�t actuelle // Copy the current forest
//
//    bool allBurnedOut = true;  // Indicateur pour v�rifier si tous les feux sont �teints // Flag to check if all fires are out
//
//    for (int i = 0; i < N; i++) {
//        for (int j = 0; j < N; j++) {
//            if (forest[i * N + j] == 2) {  // Si l'arbre est en feu // If the tree is on fire
//                burnTime[i * N + j] -= 200;  // R�duire le temps de combustion // Reduce the burning time
//
//                // V�rifier si le feu est �teint // Check if the fire is out
//                if (burnTime[i * N + j] <= 0) {
//                    newForest[i * N + j] = 3;  // Marquer l'arbre comme br�l� // Mark the tree as burned
//                }
//                else {
//                    // Propagation du feu aux voisins // Propagation of fire to neighbors
//                    if (i > 0 && forest[(i - 1) * N + j] == 1 && (rand() / (float)RAND_MAX) < spreadProbability) {
//                        newForest[(i - 1) * N + j] = 2;
//                        burnTime[(i - 1) * N + j] = BURN_DURATION;
//                    }
//                    if (i < N - 1 && forest[(i + 1) * N + j] == 1 && (rand() / (float)RAND_MAX) < spreadProbability) {
//                        newForest[(i + 1) * N + j] = 2;
//                        burnTime[(i + 1) * N + j] = BURN_DURATION;
//                    }
//                    if (j > 0 && forest[i * N + j - 1] == 1 && (rand() / (float)RAND_MAX) < spreadProbability) {
//                        newForest[i * N + j - 1] = 2;
//                        burnTime[i * N + j - 1] = BURN_DURATION;
//                    }
//                    if (j < N - 1 && forest[i * N + j + 1] == 1 && (rand() / (float)RAND_MAX) < spreadProbability) {
//                        newForest[i * N + j + 1] = 2;
//                        burnTime[i * N + j + 1] = BURN_DURATION;
//                    }
//                }
//            }
//
//            // Si un arbre br�le encore, continuer la simulation // If a tree is still burning, continue the simulation
//            if (forest[i * N + j] == 2) {
//                allBurnedOut = false;
//            }
//        }
//    }
//
//    forest = newForest;  // Mettre � jour la for�t avec la nouvelle copie // Update the forest with the new copy
//
//    if (allBurnedOut) {  // Si tous les feux sont �teints, mettre la simulation en pause // If all fires are out, pause the simulation
//        isPaused = true;
//        pauseStartTime = glutGet(GLUT_ELAPSED_TIME);
//    }
//
//    auto end = std::chrono::high_resolution_clock::now();
//    std::chrono::duration<double, std::milli> elapsed = end - start;
//    printf("Sequential updateForest took %.3f ms\n", elapsed.count());
//}
//
//void updateForest() {
//    if (isPaused) {
//        if (glutGet(GLUT_ELAPSED_TIME) - pauseStartTime >= 3000) {
//            initializeForest();
//        }
//        return;
//    }
//
//    if (useCUDA) {
//        updateForestCUDA();
//    }
//    else {
//        updateForestSequential();
//    }
//}
//
//// Function to free CUDA memory
//void cleanupCUDA() {
//    if (d_forest != nullptr) {
//        cudaFree(d_forest);
//        d_forest = nullptr;
//    }
//    if (d_burnTime != nullptr) {
//        cudaFree(d_burnTime);
//        d_burnTime = nullptr;
//    }
//    if (d_new_forest != nullptr) {
//        cudaFree(d_new_forest);
//        d_new_forest = nullptr;
//    }
//    if (d_new_burnTime != nullptr) {
//        cudaFree(d_new_burnTime);
//        d_new_burnTime = nullptr;
//    }
//}
//
//// Fonction d'affichage // Display function
//void display() {
//    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);  // Effacer le tampon de couleur et de profondeur // Clear color and depth buffer
//    glLoadIdentity();  // R�initialiser la matrice mod�le-vue // Reset the model-view matrix
//    glTranslatef(offsetX, offsetY, 0.0f);  // Appliquer le d�calage // Apply translation offset
//    glScalef(zoomLevel, zoomLevel, 1.0f);  // Appliquer le zoom // Apply zoom
//    drawForest();  // Dessiner la for�t // Draw the forest
//    glutSwapBuffers();  // �changer les tampons pour afficher l'image // Swap buffers to display the image
//}
//
//// Fonction pour animer la simulation // Function to animate the simulation
//void update(int value) {
//    updateForest();  // Mettre � jour la for�t � chaque cycle // Update the forest at each cycle
//    glutPostRedisplay();  // Demander un nouveau rendu // Request a new rendering
//    glutTimerFunc(200, update, 0);  // Programmer la prochaine mise � jour dans 200 ms // Schedule the next update in 200 ms
//}
//
//// Gestion du clavier pour zoomer/d�zoomer et r�initialiser // Keyboard handling for zooming and resetting
//void keyboard(unsigned char key, int x, int y) {
//    switch (key) {
//    case '+':
//        zoomLevel *= 1.1f;  // Augmenter le niveau de zoom // Increase zoom level
//        break;
//    case '-':
//        zoomLevel /= 1.1f;  // Diminuer le niveau de zoom // Decrease zoom level
//        if (zoomLevel < 0.1f) zoomLevel = 0.1f;
//        break;
//    case 'r':  // Touche pour r�initialiser // Reset key
//        zoomLevel = 1.0f;  // R�initialiser le zoom et le d�calage // Reset zoom and offset
//        offsetX = 0.0f;
//        offsetY = 0.0f;
//        break;
//    case 27:  // Touche �chap pour quitter // Escape key to quit
//        exit(0);
//    }
//    glutPostRedisplay();  // Redessiner la sc�ne // Redraw the scene
//}
//
//// Gestion des touches fl�ch�es pour d�placer la vue // Arrow keys handling for moving the view
//void specialKeys(int key, int x, int y) {
//    switch (key) {
//    case GLUT_KEY_UP:
//        offsetY += moveSpeed / zoomLevel;  // D�placer la vue vers le haut // Move the view up
//        break;
//    case GLUT_KEY_DOWN:
//        offsetY -= moveSpeed / zoomLevel;  // D�placer la vue vers le bas // Move the view down
//        break;
//    case GLUT_KEY_LEFT:
//        offsetX += moveSpeed / zoomLevel;  // D�placer la vue vers la gauche // Move the view left
//        break;
//    case GLUT_KEY_RIGHT:
//        offsetX -= moveSpeed / zoomLevel;  // D�placer la vue vers la droite // Move the view right
//        break;
//    }
//    glutPostRedisplay();  // Redessiner la sc�ne // Redraw the scene
//}
//
//// Gestion de la souris pour d�placer la vue // Mouse handling for moving the view
//void mouseMotion(int x, int y) {
//    if (dragging) {
//        offsetX += (x - lastMouseX) * moveSpeed / zoomLevel;  // Mettre � jour le d�calage horizontal // Update horizontal offset
//        offsetY -= (y - lastMouseY) * moveSpeed / zoomLevel;  // Mettre � jour le d�calage vertical // Update vertical offset
//        lastMouseX = x;
//        lastMouseY = y;
//        glutPostRedisplay();  // Redessiner la sc�ne // Redraw the scene
//    }
//}
//
//// Fonction pour g�rer le clic de souris // Function to handle mouse clicks
//void mouse(int button, int state, int x, int y) {
//    if (button == GLUT_LEFT_BUTTON) {  // Si le bouton gauche de la souris est enfonc� // If the left mouse button is pressed
//        if (state == GLUT_DOWN) {
//            dragging = true;
//            lastMouseX = x;
//            lastMouseY = y;
//        }
//        else {
//            dragging = false;
//        }
//    }
//}
//
//// Fonction principale // Main function
//int main(int argc, char** argv) {
//    srand(static_cast<unsigned>(time(NULL)));  // Initialiser le g�n�rateur de nombres al�atoires // Initialize random number generator
//    glutInit(&argc, argv);
//    glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
//    glutInitWindowSize(800, 800);
//    glutCreateWindow("Simulation de feux de for�t/Forest Fire Simulation");  // Cr�er la fen�tre OpenGL // Create the OpenGL window
//
//    initGL();
//    initializeForest();
//
//    glutDisplayFunc(display);
//    glutKeyboardFunc(keyboard);
//    glutSpecialFunc(specialKeys);
//    glutMouseFunc(mouse);
//    glutMotionFunc(mouseMotion);
//    glutTimerFunc(200, update, 0);
//
//    glutMainLoop();
//    cleanupCUDA();
//    return 0;
//}
